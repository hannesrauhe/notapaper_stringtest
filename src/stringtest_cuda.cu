#include "stringtest.hpp"

void gpureset() {
    hipDeviceReset();
}

void gpuStringArray::init(std::vector<std::string>& haystack, int noChar) {
//    boost::timer::auto_cpu_timer t;
    size = haystack.size();
    byte_size = noChar + haystack.size()*sizeof(uint)*2;
    CUDA_CHECK_RETURN(hipHostAlloc(&length,haystack.size()*sizeof(uint),hipHostMallocDefault));
    CUDA_CHECK_RETURN(hipHostAlloc(&pos,haystack.size()*sizeof(uint),hipHostMallocDefault));
    //space for last terminating character because it's easier
    CUDA_CHECK_RETURN(hipHostAlloc(&data,(1+noChar)*sizeof(char),hipHostMallocDefault));
    CUDA_CHECK_RETURN(hipHostAlloc(&first,haystack.size()*sizeof(char),hipHostMallocDefault));

    int cpos = 0;
    for(int i = 0;i<size;++i) {
        pos[i]=cpos;
        length[i]=haystack[i].length();
        first[i]=haystack[i][0];
        strcpy(data+cpos, haystack[i].c_str());
        cpos+=haystack[i].length();
    }
}

void gpuStringArray::destroy() {
    byte_size = 0;
    CUDA_CHECK_RETURN(hipHostFree(data));
    CUDA_CHECK_RETURN(hipHostFree(pos));
    CUDA_CHECK_RETURN(hipHostFree(length));
    CUDA_CHECK_RETURN(hipHostFree(first));
}
